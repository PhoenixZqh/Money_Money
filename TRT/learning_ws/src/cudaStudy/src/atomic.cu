#include <hip/hip_runtime.h>

#include <iostream>
#include <stdio.h>
#include <string.h>

/*
    对同一块内存进行操作时，需要用到原子操作，不然就是乱序的
*/

__global__ void AtomicAdd(int* d_in)
{
    int Idx = blockDim.x * blockIdx.x + threadIdx.x;

    // atomicAdd(&d_in[Idx], 1);
    int test = atomicAdd(d_in, 1);
    printf("11111:%d", test);

    __syncthreads();

    printf("11111:%d", test);

    // d_in[Idx] += 1;
}

int main(int argc, char** argv)
{

    const int arrSize = 10;
    const int arrBytes = arrSize * sizeof(int);

    int h_in[arrSize];
    int h_out[arrSize];
    int* d_in = nullptr;

    for (int i = 0; i < arrSize; i++) {
        h_in[i] = i;
    }

    hipMalloc((void**)&d_in, arrBytes); // 注意这里取地址

    hipMemcpy(d_in, h_in, arrBytes, hipMemcpyHostToDevice);

    int b = 10;
    int* a = &b;
    AtomicAdd<<<10, 100>>>(a);

    hipMemcpy(h_out, d_in, arrBytes, hipMemcpyDeviceToHost);

    for (int i = 0; i < arrSize; i++) {
        printf("h_out[%d]:%d\n", i, h_out[i]);
    }

    hipFree(d_in);

    hipDeviceReset();
    return 0;
}