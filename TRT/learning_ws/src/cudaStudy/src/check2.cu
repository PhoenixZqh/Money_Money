#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

/*
1. 采用宏定义在每次调用API前都检查初始化
2. 采用封装带参宏定义使代码更加清晰、好调试
*/

#define checkDriver(op) __check_cuda_driver((op), #op, __FILE__, __LINE__) // #op 加一个#是变成字符串的意思

bool __check_cuda_driver(hipError_t code, const char* op, const char* file, int line)
{
    if (code != hipError_t::hipSuccess) {
        const char* err_name = nullptr;
        const char* err_msg = nullptr;
        hipDrvGetErrorName(code, &err_name);
        hipDrvGetErrorString(code, &err_msg);
        printf("%s:%d  %s failed. \n  code = %s, message = %s\n", file, line, op, err_name, err_msg);
        return false;
    }

    return true;
}

int main(int argc, char** argv)
{

    checkDriver(hipInit(0));

    // 测试获取当前cuda驱动的版本
    int driver_version = 0;
    if (!checkDriver(hipDriverGetVersion(&driver_version))) {
        return -1;
    }
    printf("Driver version is %d\n", driver_version);

    // 测试获取当前设备信息
    char device_name[100];
    hipDevice_t device = 0;
    checkDriver(hipDeviceGetName(device_name, sizeof(device_name), device));
    printf("Device %d name is %s\n", device, device_name);

    return 0;
}