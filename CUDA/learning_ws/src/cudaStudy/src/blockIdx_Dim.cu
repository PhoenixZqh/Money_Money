#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void checkIndex()
{
    printf("threadIdx:(%d,%d,%d) blockIdx:(%d,%d,%d) blockDim:(%d,%d,%d)\
  gridDim(%d,%d,%d)\n",
        threadIdx.x, threadIdx.y, threadIdx.z,
        blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z,
        gridDim.x, gridDim.y, gridDim.z);
}

int main(int argc, char** argv)
{
    int nElem = 6;
    dim3 block(3);
    dim3 grid((nElem + block.x - 1) / block.x); // dim 是对应的结构体来保存idx中三个字段的范围的

    // dim3是手工定义的，主机端可见。uint3是设备端在执行的时候可见的，不可以在核函数运行时修改，初始化完成后uint3值就不变了。
    printf("grid.x %d grid.y %d grid.z %d\n", grid.x, grid.y, grid.z);
    printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);

    checkIndex<<<grid, block>>>();
    hipDeviceReset();
    return 0;
}