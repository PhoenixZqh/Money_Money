#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <string.h>

using namespace std;

__global__ void hello_world(void) //__global__ 告诉编译器这个是个可以在设备上执行的核函数
{

    printf("GPU: Hello world!\n");
}

int main(int argc, char** argv)
{

    printf("CPU: Hello world11111111111111111!\n");

    hello_world<<<2, 10>>>(); //执行2个10次
    hipDeviceReset(); //功能相当于告诉主机线程，等GPU上的程序运行完了之后再退出

    return 0;
}