#include "hip/hip_runtime.h"
#include <cudaStudy/cudaHelper.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printThreadIndex(float* A, const int nx, const int ny)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix; // ix ,iy 是线程模型中的全局地址

    printf("thread_id(%d,%d) block_id(%d,%d) coordinate(%d,%d)"
           "global index %2d ival %2d\n",
        threadIdx.x, threadIdx.y,
        blockIdx.x, blockIdx.y, ix, iy, idx, A[idx]);
}

int main(int argc, char** argv)
{
    initDevice(0);
    int nx = 8, ny = 6;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);

    float* A_h = (float*)malloc(nBytes);
    initialData(A_h, nxy);
    printMatrix(A_h, nx, ny);

    float* A_d = NULL;
    CHECK(hipMalloc((void**)&A_d, nBytes));
    hipMemcpy(A_d, A_h, nBytes, hipMemcpyHostToDevice);

    dim3 block(4, 2);
    dim3 grid((nx - 1) / block.x + 1, (ny - 1) / block.y + 1);
    printThreadIndex<<<grid, block>>>(A_d, nx, ny); // <<<gridDim(有多少个block), blockDim(有多少个thread)>>>

    CHECK(hipDeviceSynchronize());
    hipFree(A_d);
    free(A_h);

    hipDeviceReset();
    return 0;
}
