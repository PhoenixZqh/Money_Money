#include "hip/hip_runtime.h"
#include <cudaStudy/cudaHelper.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
两个向量的加法
一个核函数只有一个grid， 一个grid有多个block， 一个block有多个线程
*/

// 在cpu上运行
void sumArrays(float* a, float* b, float* res, const int size)
{
    for (int i = 0; i < size; i += 4) {
        res[i] = a[i] + b[i];
        res[i + 1] = a[i + 1] + b[i + 1];
        res[i + 2] = a[i + 2] + b[i + 2];
        res[i + 3] = a[i + 3] + b[i + 3];
    }
}

// 核函数在GPU上运行
__global__ void sumArraysGpu(float* a, float* b, float* res)
{
    int i = threadIdx.x;
    res[i] = a[i] + b[i];
}

int main(int argc, char** argv)
{
    int dev = 0;
    hipSetDevice(dev); // 设置想利用的设备，内存分配和核函数的启动都会在当前设备上

    int nElem = 32;
    printf("Vector size:%d\n", nElem);

    int nByte = sizeof(float) * nElem;
    float* a_h = (float*)malloc(nByte);
    float* b_h = (float*)malloc(nByte);
    float* res_h = (float*)malloc(nByte);
    float* res_gpu_h = (float*)malloc(nByte);

    // memset 用于特定值填充内存块
    memset(res_h, 0, nByte);
    memset(res_gpu_h, 0, nByte);

    float *a_d, *b_d, *res_d;
    // cudamalloc 返回值是CUDA中定义的一个错误代码
    CHECK(hipMalloc((float**)&a_d, nByte)); // 分配设备端的内存空间, 指针是一个变量，既然是变量，就需要存储空间，存储的是指针变量的地址(想象void*),之后向这个地址中写入一个地址值（在GPU显存中）
    CHECK(hipMalloc((float**)&b_d, nByte));
    CHECK(hipMalloc((float**)&res_d, nByte));

    initialData(a_h, nElem);
    initialData(b_h, nElem);

    // hipMemcpy用于在主机（Host）和设备（Device）之间往返的传递数据;   该函数是同步执行函数
    CHECK(hipMemcpy(a_d, a_h, nByte, hipMemcpyHostToDevice)); // dst ，src
    CHECK(hipMemcpy(b_d, b_h, nByte, hipMemcpyHostToDevice));

    dim3 block(nElem);
    dim3 grid(nElem / block.x);

    double iStart, iElaps;
    iStart = cpuSecond();
    sumArraysGpu<<<grid, block>>>(a_d, b_d, res_d);
    hipDeviceSynchronize(); // 加上同步函数才是核函数的执行时间

    CHECK(hipMemcpy(res_gpu_h, res_d, nByte, hipMemcpyDeviceToHost));
    iElaps = cpuSecond() - iStart;
    printf("Execution configuration<<<%d,%d>>> Time elapsed %f sec\n", grid.x, block.x, iElaps);

    sumArrays(a_h, b_h, res_h, nElem);

    checkResult(res_h, res_gpu_h, nElem);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(res_d);

    free(a_h);
    free(b_h);
    free(res_h);
    free(res_gpu_h);

    return 0;
}