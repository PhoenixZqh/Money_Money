#include <hip/hip_runtime.h>

#include <stdio.h>
#include <string.h>

/*
1. 通过有参宏定义检查cuda driver 是否正常初始化、定位程序出错文件名、行数、以及错误信息
2. 宏定义中带有do while 保证程序的正确性
*/

#define checkDriver(op)                                                                                            \
    do {                                                                                                           \
        auto code = (op);                                                                                          \
        if (code != hipError_t::hipSuccess) {                                                                      \
            const char* err_name = nullptr;                                                                        \
            const char* err_msg = nullptr;                                                                         \
            hipDrvGetErrorName(code, &err_name);                                                                       \
            hipDrvGetErrorString(code, &err_msg);                                                                      \
            printf("%s:%d  %s failed. \n  code = %s, message = %s\n", __FILE__, __LINE__, #op, err_name, err_msg); \
            return -1;                                                                                             \
        }                                                                                                          \
    } while (0)

int main(int argc, char** argv)
{

    hipInit(2); // 初始化flag设置为0
    checkDriver(hipInit(0)); // cuda driver需要做显式的初始化cuInit(0), 不然其他API返回CUDA_ERROR_NOT_INITIALIZED

    int driverVersion = 0;
    checkDriver(hipDriverGetVersion(&driverVersion)); // 测试cuda版本
    printf("Driver version is %d\n", driverVersion);

    char deviceName[100];
    hipDevice_t device = 0;
    checkDriver(hipDeviceGetName(deviceName, sizeof(deviceName), device)); // 测试当前设备信息
    printf("Device %d name is %s\n", device, deviceName);

    return 0;
}