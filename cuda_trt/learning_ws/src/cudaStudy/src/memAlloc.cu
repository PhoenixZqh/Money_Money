#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

/*

*/

#define checkDriver(op) __check_cuda_driver((op), #op, __FILE__, __LINE__)

bool __check_cuda_driver(hipError_t code, const char* op, const char* file, int line)
{
    if (code != hipError_t::hipSuccess) {
        const char* err_name = nullptr;
        const char* err_msg = nullptr;
        hipDrvGetErrorName(code, &err_name);
        hipDrvGetErrorString(code, &err_msg);
        printf("%s:%d  %s failed. \n  code = %s, message = %s\n", file, line, op, err_name, err_msg);

        return false;
    }

    return true;
}

int main(int argc, char** argv)
{

    checkDriver(hipInit(0));

    // 创建上下文
    hipCtx_t context = nullptr;
    hipDevice_t device = 0;
    checkDriver(hipCtxCreate(&context, hipDeviceScheduleAuto, device));
    printf("context=%p\n", context);

    // 向设备要一个100字节大小的线性内存，并返回其地址
    hipDeviceptr_t device_memPtr = 0;
    checkDriver(hipMalloc(&device_memPtr, 100));
    printf("device memory Pointer=%p\n", device_memPtr);

    // 二级指针向host要一个100字节大小的锁页内存
    float* host_page_lock_memPtr = nullptr;
    checkDriver(hipMemAllocHost((void**)&host_page_lock_memPtr, 100));
    printf("host_page_lock_memPtr=%p\n", host_page_lock_memPtr);

    // 向分页内存上放数据（在cpu上）,可以让GPU快速读取
    host_page_lock_memPtr[0] = 111;
    printf("host_page_lock_memPtr[0]=%f\n", host_page_lock_memPtr[0]);

    float new_val = 222;
    checkDriver(hipMemsetD32((hipDeviceptr_t)host_page_lock_memPtr, *(int*)&new_val, 1));
    printf("after change,host_page_lock_memPtr[0]=%f\n", host_page_lock_memPtr[0]);

    // 释放内存
    checkDriver(hipHostFree(host_page_lock_memPtr));

    hipDeviceReset();
    return 0;
}