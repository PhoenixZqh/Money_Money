#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <string.h>

/*

*/

using namespace std;

// 在全局内存上操作
__global__ void global_reduce_kernel(float* d_in, float* d_out)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x; // 所有线程放在一起你是第几号
    int tId = threadIdx.x; // 在当前block中你是第几号

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tId < s) {
            d_in[myId] += d_in[myId + s];
        }

        __syncthreads(); // 等待所有线程完成任务
    }

    if (tId == 0) {
        d_out[blockIdx.x] = d_in[myId]; // 第一个block的总和等于该block第一个线程的值，最后对折到只剩一个线程
    }

    // printf("gpu res:%f\n", d_out);
}

// 在共享内存上操作
__global__ void shared_reduce_kernel(float* d_in, float* d_out)
{
    extern __shared__ float data[]; // 创建共享内存

    int myId = threadIdx.x + blockDim.x * blockIdx.x; // 全局的线程索引？
    int tId = threadIdx.x; // 线程号

    data[tId] = d_in[myId];
    __syncthreads();

    for (unsigned int s = blockIdx.x / 2; s > 0; s >>= 1) {
        if (tId < s) {
            data[tId] += d_in[tId + s];
        }
        __syncthreads();
    }

    if (tId == 0) {
        d_out[blockIdx.x] = data[0];
    }
}

void reduce(float* d_in, float* d_intermediate, float* d_out, int size, bool useSharedMem)
{
    const int maxPerBlock = 1024;
    int threads = maxPerBlock;
    int block = size / maxPerBlock;

    if (useSharedMem) {
        shared_reduce_kernel<<<block, threads, threads * sizeof(float)>>>(d_in, d_intermediate);
    } else {
        global_reduce_kernel<<<block, threads>>>(d_in, d_intermediate);
    }

    // 将所有block的值相加
    threads = block;
    block = 1;

    if (useSharedMem) {
        shared_reduce_kernel<<<block, threads, threads * sizeof(float)>>>(d_intermediate, d_out);
    } else {
        global_reduce_kernel<<<block, threads>>>(d_intermediate, d_out);
    }
}

int main(int argc, char** argv)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        fprintf(stderr, "err: no device supprot cuda");
        exit(EXIT_FAILURE);
    }

    int dev = 0;
    hipSetDevice(dev);

    hipDeviceProp_t devProps;
    if (hipGetDeviceProperties(&devProps, dev) == 0) {
        printf("Using device % d \n", dev);
        printf("%s;global mem: %dB;compute v%d.%d;clock:%dkHZ\n",
            devProps.name, (int)devProps.totalGlobalMem, (int)devProps.major, (int)devProps.minor, (int)devProps.clockRate);
    }

    const int arrSize = 1 << 20;
    const int arrBytes = arrSize * sizeof(float);

    float h_in[arrSize];
    float sum = 0.0f;

    for (int i = 0; i < arrSize; i++) {
        // h_in[i] = -1.0f + (float)random() / ((float)RAND_MAX / 2.0f);
        h_in[i] = (i + 1) * 0.1;
        // printf("h_in:%f\n", h_in[i]);
        sum += h_in[i];
    }

    printf("sum:%f\n", sum);

    float* d_in;
    float* d_intermediate;
    float* d_out;

    hipMalloc((void**)&d_in, arrBytes);
    hipMalloc((void**)&d_intermediate, arrBytes);
    hipMalloc((void**)&d_out, sizeof(float));

    hipMemcpy(d_in, h_in, arrBytes, hipMemcpyHostToDevice);
    int which_kernel = 0;
    if (argc == 2)
        which_kernel = atoi(argv[1]);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    switch (which_kernel) {
    case 0:
        printf("running gloabal reduce\n");
        hipEventRecord(start, 0);
        // for (int i = 0; i < 100; i++) {
        cout << "d_in " << h_in[0] << endl;

        reduce(d_in, d_intermediate, d_out, arrSize, false);
        // }

        hipEventRecord(stop, 0);
        break;

    case 1:
        printf("running shared reduce\n");
        hipEventRecord(start, 0);
        // for (int i = 0; i < 100; i++) {
        reduce(d_in, d_intermediate, d_out, arrSize, true);
        // }

        hipEventRecord(stop, 0);
        break;

    default:
        fprintf(stderr, "err:ran no kernel\n");
        exit(EXIT_FAILURE);
    }

    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    elapsedTime /= 100.f;

    float h_out;
    hipMemcpy(&h_out, d_out, sizeof(float), hipMemcpyDeviceToHost); // 该函数是同步执行函数，在未完成数据的转移操作之前会锁死并一直占有CPU进程的控制权，所以不用再添加hipDeviceSynchronize()函数

    printf("all:%f\n", h_out);

    printf("average time elapsed:%f\n", elapsedTime);

    hipFree(d_in);
    hipFree(d_intermediate);
    hipFree(d_out);

    return 0;
}