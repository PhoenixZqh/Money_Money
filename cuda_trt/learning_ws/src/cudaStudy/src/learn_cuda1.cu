
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <string.h>

/*

*/
using namespace std;

__global__ void square(float* d_in, float* d_out)
{
    int idx = threadIdx.x; //.x表示当前线程号

    float f = d_in[idx];
    d_out[idx] = f * f * f;
}

int main(int argc, char** argv)
{

    const int arr_size = 8;
    const int arr_bytes = arr_size * sizeof(float);

    float h_in[arr_size];

    for (int i = 0; i < arr_size; i++) {
        h_in[i] = float(i);
    }

    float h_out[arr_size];

    float* d_in;
    float* d_out;

    hipMalloc((void**)&d_in, arr_bytes);
    hipMalloc((void**)&d_out, arr_bytes);

    hipMemcpy(d_in, h_in, arr_bytes, hipMemcpyHostToDevice);

    square<<<1, 8>>>(d_in, d_out);

    hipMemcpy(h_out, d_out, arr_bytes, hipMemcpyDeviceToHost);

    for (int i = 0; i < arr_size; i++) {
        cout << h_out[i] << endl;
    }

    hipFree(d_in);
    hipFree(d_out);

    return 0;
}