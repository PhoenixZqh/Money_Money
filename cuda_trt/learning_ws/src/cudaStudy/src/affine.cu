#include <hip/hip_runtime.h>

#include <iostream>
#include <string.h>

/*
实现仿射变换（旋转、平移、缩放）
1. 根据src的尺寸、dst的尺寸计算src 到 dst 的矩阵关系 I2D
2. 根据I2D矩阵求逆可以得到dst 到 src 的矩阵关系 D2I
3. 根据D2I， 可以求得，dst上当前点在src上的位置
4. 位置已知，可以根据双线性插值求得该点的值
*/

#define min(a, b) ((a) < (b) ? (a) : (b))
#define num_threads 512

typedef unsigned char uint8_t;
struct Size {
    int m_width = 0;
    int m_height = 0;

    Size() = default;
    Size(int w, int h)
        : m_width(w)
        , m_height(h)
    {
    }
};

struct AffineMatrix {

    float I2D[6];
    float D2I[6];

    /*********************************************求逆矩阵**************************************************/
    void InvertAffineTransform(float imat[6], float omat[6])
    {
        float i00 = imat[0];
        float i01 = imat[1];
        float i02 = imat[2];
        float i10 = imat[3];
        float i11 = imat[4];
        float i12 = imat[5];

        // 计算行列式
        float D = i00 * i11 - i01 * i10;
        D = D != 0 ? 1.0 / D : 0;

        // 计算剩余的伴随矩阵除以行列式
        float A11 = i11 * D;
        float A22 = i00 * D;
        float A12 = -i01 * D;
        float A21 = -i10 * D;
        float b1 = -A11 * i02 - A12 * i12;
        float b2 = -A21 * i02 - A22 * i12;
        omat[0] = A11;
        omat[1] = A12;
        omat[2] = b1;
        omat[3] = A21;
        omat[4] = A22;
        omat[5] = b2;
    }

    /*********************************************计算I2D*************************************************/
    void ComputeI2D(const Size& from, const Size& to)
    {
        float scale_x = to.m_width / (float)from.m_width;
        float scale_y = to.m_height / (float)from.m_height;

        float scale = min(scale_x, scale_y); // 不能取最大，防止越界

        I2D[0] = scale;
        I2D[1] = 0;
        I2D[2] = -scale * from.m_width * 0.5 + to.m_width * 0.5 - 0.5;

        I2D[3] = 0;
        I2D[4] = scale;
        I2D[5] = -scale * from.m_height * 0.5 + to.m_height * 0.5 + scale * 0.5 - 0.5;

        InvertAffineTransform(I2D, D2I); // 求解逆矩阵是为了找到dst对应src上的位置
    }
};

/*********************************************计算dst上的点对应在src上的位置*****************************/
__device__ void FindPosSrc(float* matrix, int x, int y, float* src_x, float* src_y)
{
    *src_x = matrix[0] * x + matrix[1] * y + matrix[2];
    *src_y = matrix[3] * x + matrix[4] * y + matrix[5];
}

/*********************************************双线性插值确认这个点的值************************************/
__global__ void Warp_Affine_Bilinear_Kernel(
    uint8_t* src, int src_line_size, int src_width, int src_height,
    uint8_t* dst, int dst_line_size, int dst_width, int dst_height,
    uint8_t fill_value, AffineMatrix matrix)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x; // 两个维度上的索引
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idx >= dst_width || idy >= dst_height) // 超出边界
        return;

    float c0 = fill_value, c1 = fill_value, c2 = fill_value;
    float src_x = 0;
    float src_y = 0;
    FindPosSrc(matrix.D2I, idx, idy, &src_x, &src_y);

    if (src_x < -1 || src_x >= src_width || src_y < -1 || src_y >= src_height) {

        // 这里我的理解是，该像素点周围的四个点 (x_low,y_low),(x_high,y_low),(x_low,y_high),(x_high,y_high)
        // 如果src_x < -1, x_low取整为-1,超出边界，要保证x_low最小为0； y方向上是一致的
    }

    else {
        int y_low = floorf(src_y);
        int x_low = floorf(src_x);
        int y_high = y_low + 1;
        int x_high = x_low + 1;

        uint8_t values[] = { fill_value, fill_value, fill_value }; // 超出边界的填充值

        float lx = src_x - x_low;
        float ly = src_y - y_low;
        float hx = 1 - lx;
        float hy = 1 - ly;

        float w1 = hx * hy;
        float w2 = lx * hy;
        float w3 = hx * ly;
        float w4 = lx * ly;

        uint8_t* v1 = values; // 指针指向的是地址，这里是给周围的四个点赋初值
        uint8_t* v2 = values;
        uint8_t* v3 = values;
        uint8_t* v4 = values;

        if (y_low >= 0) {
            if (x_low >= 0) {
                v1 = src + y_low * src_line_size + x_low * 3; // src_line_size表示图像宽度x3；
            }

            if (x_high < src_width) {
                v2 = src + y_low * src_line_size + x_high * 3;
            }
        }

        if (y_high < src_height) {
            if (x_low >= 0) {
                v3 = src + y_high * src_line_size + x_low * 3;
            }

            if (x_low < src_width) {
                v4 = src + y_high * src_line_size + x_high * 3;
            }
        }

        c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
        c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
        c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);

        uint8_t* pdst = dst + idy * dst_line_size + idx * 3;
        pdst[0] = c0;
        pdst[1] = c1;
        pdst[2] = c2;
    }
}

void WarpAffineBilinear(
    uint8_t* src, int src_line_size, int src_width, int src_height,
    uint8_t* dst, int dst_line_size, int dst_width, int dst_height,
    uint8_t fill_value)
{
    dim3 block_size(32, 32);
    dim3 grid_size((dst_width + 31) / 32, (dst_height + 31) / 32);

    AffineMatrix affine;
    affine.ComputeI2D(Size(src_width, src_height), Size(dst_width, dst_height));

    Warp_Affine_Bilinear_Kernel<<<grid_size, block_size, 0, nullptr>>>( //<<<gridsize,blocksize,共享内存字节数，流>>>
        src, src_line_size, src_width, src_height,
        dst, dst_line_size, dst_width, dst_height,
        fill_value, affine);
}
