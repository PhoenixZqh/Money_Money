#include <hip/hip_runtime.h>

#include <iostream>
#include <stdio.h>
#include <string.h>

/*
1. 扫描并行化
2. 0,1,2,3,4,5,6,7 ,累计相加等于28
3. 算法的思想是以2的次方逐步增加
*/

__global__ void globalScan(float* d_in, float* d_out)
{
    int idx = threadIdx.x; // 获取线程ID号，每个线程代表一个数
    float out = 0.00f;

    d_out[idx] = d_in[idx]; // 先将输入数组复制给输出数组
    __syncthreads(); // 等所有的线程都完成工作

    // printf("sizeof d_In: %d\n", sizeof(d_in));

    for (int i = 1; i < sizeof(d_in); i *= 2) {
        if (idx - i >= 0) {
            out = d_out[idx] + d_out[idx - i];
        }
        __syncthreads();

        if (idx - i >= 0) {
            d_out[idx] = out;
            out = 0.00f;
        }

        __syncthreads();
        // printf("d_out:%f\n", d_out[idx]);
    }
    // printf("all:%f\n", d_out[7]);
}

int main(int argc, char** argv)
{
    const int arrSize = 8;
    const int arrBytes = arrSize * sizeof(float);

    float h_in[arrSize];
    float h_out[arrSize];

    for (int i = 0; i < arrSize; i++) {
        h_in[i] = float(i);
        // printf("h_in[%d]:%f\n", i, h_in[i]);
    }

    float* d_in;
    float* d_out;
    hipMalloc((void**)&d_in, arrBytes);
    hipMalloc((void**)&d_out, arrBytes); // 在GPU上分配内存空间

    hipMemcpy(d_in, h_in, arrBytes, hipMemcpyHostToDevice);

    globalScan<<<1, arrSize>>>(d_in, d_out);

    hipMemcpy(h_out, d_out, arrBytes, hipMemcpyDeviceToHost);

    for (int i = 0; i < arrSize; i++) {
        printf("[%d]得到的结果:%f\n", i, h_out[i]);
    }

    // cudaDeviceReset();
    return 0;
}