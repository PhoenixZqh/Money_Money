#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

/*
1. 什么是上下文？
    设备与特定进程相关连的所有状态称之为上下文；比如写了一段核函数对GPU的使用造成不同的状态（内存映射、分配、加载的code），
    上下文则保存所有的管理数据来控制和使用设备；
2. 上下文管理可以干哪些工作？
    a. 持有分配的内存列表
    b. 持有加载进该设备的kernel code
    c. cpu和GPU之间的unified memory

3. 如何管理上下文？
    a. 需手动开关，开始时hipCtxCreate()创建上下文，结束时hipCtxDestroy销毁上下文；
    b. 用hipDevicePrimaryCtxRetain()创建上下文更好
    c. hipCtxGetCurrent()获取当前上下文
    d. 可以使用堆栈管理多个上下文hipCtxPushCurrent()压入，hipCtxPopCurrent()推出
    e. 对ctxA使用hipCtxPushCurrent()和hipCtxCreate()都相当于将ctxA放到栈顶（让它成为current context）
*/

#define checkDriver(op) __check_cuda_driver((op), #op, __FILE__, __LINE__)

bool __check_cuda_driver(hipError_t code, const char* op, const char* file, int line)
{
    if (code != hipError_t::hipSuccess) {
        const char* err_name = nullptr;
        const char* err_msg = nullptr;
        hipDrvGetErrorName(code, &err_name);
        hipDrvGetErrorString(code, &err_msg);
        printf("%s:%d  %s failed. \n  code = %s, message = %s\n", file, line, op, err_name, err_msg);
        return false;
    }

    return true;
}

int main(int argc, char** argv)
{
    checkDriver(hipInit(0));

    // 为设备创建上下文
    hipCtx_t ctxA = nullptr;
    hipCtx_t ctxB = nullptr;

    hipDevice_t device = 0;

    // 告诉某一块设备要在哪个地方创建ctxA管理数据
    checkDriver(hipCtxCreate(&ctxA, hipDeviceScheduleAuto, device));
    checkDriver(hipCtxCreate(&ctxB, hipDeviceScheduleAuto, device));
    printf("ctxA = %p\n", ctxA);
    printf("ctxB = %p\n", ctxB);

    // 获取当前上下文信息
    hipCtx_t current_context = nullptr;
    checkDriver(hipCtxGetCurrent(&current_context));
    printf("current_context = %p\n", current_context);

    // 使用上下文堆栈对设备管理多个上下文
    checkDriver(hipCtxPushCurrent(ctxA)); // 将这个 ctxA 压入CPU调用的thread上。专门用一个thread以栈的方式来管理多个contexts的切换
    checkDriver(hipCtxGetCurrent(&current_context)); // 获取current_context (即栈顶的context)
    printf("after pushing, current_context = %p\n", current_context);

    // 弹出当前content
    hipCtx_t popped_ctx = nullptr;
    checkDriver(hipCtxPopCurrent(&popped_ctx));
    checkDriver(hipCtxGetCurrent(&current_context));
    printf("after poping, popped_ctx = %p\n", popped_ctx); // 弹出的是ctxA
    printf("after poping, current_context = %p\n", current_context); // current_context是ctxB

    // 销毁上下文
    checkDriver(hipCtxDestroy(ctxA));
    checkDriver(hipCtxDestroy(ctxB));

    checkDriver(hipDevicePrimaryCtxRetain(&ctxA, device)); // 在 device 上指定一个新地址对ctxA进行管理
    printf("ctxA = %p\n", ctxA);
    checkDriver(hipDevicePrimaryCtxRelease(device));

    return 0;
}